
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define N 20480

// declare the kernel
__global__ void daxpy(double a, double *x, double *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        y[i] += a*x[i];
    }
}

int main(void) {
    double *x, *y, a;
    double m = -1.;
    double tmp;
    int i;
    size_t size = N*sizeof(double);

    // allocate unified memory
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);

    // initialize x and y
    srand(time(NULL));
    a = (double)random() / RAND_MAX;
    for (i=0; i<N; i++)
        x[i] = (double)random() / RAND_MAX;
    for (i=0; i<N; i++)
        y[i] = (double)random() / RAND_MAX;

    // launch the kernel function
    daxpy<<<N/256,256>>>(a, x, y);

    hipDeviceSynchronize();

    // deallocate device memory
    hipFree(x);
    hipFree(y);
}
